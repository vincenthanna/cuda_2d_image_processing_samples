#include "hip/hip_runtime.h"
#include <vector>
#include <tuple>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <chrono>
#include <iostream>

// kernel functions :
__global__ void drawBox(unsigned char* d_image, int x, int y, int patchWidth, int patchHeight, int width, int height, int channels) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix > x && ix < (x + patchWidth) && iy > y && iy < (y + patchHeight) && iz < channels) {
        int dIndex = (iy * width + ix) * channels + iz;
        d_image[dIndex] = 0;
    }
}

void draw_box(unsigned char* buffer, int width, int height, int channels, std::vector<std::tuple<int, int, int, int>>& xyxys) {
    int imageSize = width * height * channels * sizeof(unsigned char);
    unsigned char* d_image;
    // allocate device memory
    hipMalloc(&d_image, imageSize);
    // copy image from host to device
    hipMemcpy(d_image, buffer, imageSize, hipMemcpyHostToDevice);

    for (int i = 0; i < xyxys.size(); i++) {
        // get start time in milliseconds
        auto start = std::chrono::high_resolution_clock::now();

        int x = std::get<0>(xyxys[i]);
        int y = std::get<1>(xyxys[i]);
        int patchWidth = std::get<2>(xyxys[i]) - std::get<0>(xyxys[i]);
        int patchHeight = std::get<3>(xyxys[i]) - std::get<1>(xyxys[i]);

        dim3 blockDim(16, 16, channels);
        dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
        drawBox<<<gridDim, blockDim>>>(d_image, x, y, patchWidth, patchHeight, width, height, channels);

        // get end time in milliseconds
        auto end = std::chrono::high_resolution_clock::now();
        // print elapsed time in milliseconds
        std::cout << "Elapsed time in milliseconds : " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms" << std::endl;
    }

    // Copy image from device to host
    hipMemcpy(buffer, d_image, imageSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_image);

    hipDeviceSynchronize();
}